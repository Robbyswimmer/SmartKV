#include "hip/hip_runtime.h"
/*
 * SmartKV CUDA Kernels - Implementation
 *
 * Fused quantized attention with on-the-fly dequantization.
 * Phase 2.2: Minimal kernel implementation with numerically stable softmax
 */

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cfloat>
#include <cstdint>

namespace {

// Warp-level reduction for max/sum
__device__ __forceinline__ float warp_reduce_max(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Block-level reduction for max
__device__ __forceinline__ float block_reduce_max(float val, float* shared) {
    int lane = threadIdx.x % 32;
    int wid = threadIdx.x / 32;

    // Warp-level reduction
    val = warp_reduce_max(val);

    // Write to shared memory
    if (lane == 0) shared[wid] = val;
    __syncthreads();

    // Final reduction across warps
    if (wid == 0) {
        val = (lane < (blockDim.x / 32)) ? shared[lane] : -FLT_MAX;
        val = warp_reduce_max(val);
    }

    return val;
}

// Block-level reduction for sum
__device__ __forceinline__ float block_reduce_sum(float val, float* shared) {
    int lane = threadIdx.x % 32;
    int wid = threadIdx.x / 32;

    // Warp-level reduction
    val = warp_reduce_sum(val);

    // Write to shared memory
    if (lane == 0) shared[wid] = val;
    __syncthreads();

    // Final reduction across warps
    if (wid == 0) {
        val = (lane < (blockDim.x / 32)) ? shared[lane] : 0.0f;
        val = warp_reduce_sum(val);
    }

    return val;
}

// ============================================================================
// On-the-fly unpacking helpers for bucket-aware kernel
// ============================================================================

// Unpack a single 2-bit value from packed byte
__device__ __forceinline__ int8_t unpack_2bit_value(const uint8_t* packed, int idx) {
    int byte_idx = idx / 4;
    int bit_offset = (idx % 4) * 2;
    uint8_t packed_byte = packed[byte_idx];
    uint8_t unsigned_val = (packed_byte >> bit_offset) & 0x3;
    return (int8_t)(unsigned_val - 2);  // [0,3] -> [-2,1]
}

// Unpack a single 3-bit value from packed bytes
__device__ __forceinline__ int8_t unpack_3bit_value(const uint8_t* packed, int idx) {
    int bit_offset = idx * 3;
    int byte_idx = bit_offset / 8;
    int bit_in_byte = bit_offset % 8;

    // Read up to 2 bytes to get 3 bits
    uint32_t window = 0;
    window = packed[byte_idx];
    if (bit_in_byte + 3 > 8) {
        window |= ((uint32_t)packed[byte_idx + 1]) << 8;
    }

    uint8_t unsigned_val = (window >> bit_in_byte) & 0x7;
    return (int8_t)(unsigned_val - 4);  // [0,7] -> [-4,3]
}

// Unpack a single 4-bit value from packed byte
__device__ __forceinline__ int8_t unpack_4bit_value(const uint8_t* packed, int idx) {
    int byte_idx = idx / 2;
    int bit_offset = (idx % 2) * 4;
    uint8_t packed_byte = packed[byte_idx];
    uint8_t unsigned_val = (packed_byte >> bit_offset) & 0xF;
    return (int8_t)(unsigned_val - 8);  // [0,15] -> [-8,7]
}

// Generic unpacking dispatcher (compile-time template)
template<int BITS>
__device__ __forceinline__ int8_t unpack_value(const void* packed_ptr, int idx) {
    const uint8_t* packed = reinterpret_cast<const uint8_t*>(packed_ptr);
    if (BITS == 2) {
        return unpack_2bit_value(packed, idx);
    } else if (BITS == 3) {
        return unpack_3bit_value(packed, idx);
    } else if (BITS == 4) {
        return unpack_4bit_value(packed, idx);
    } else {
        // 8-bit: direct load as int8
        return reinterpret_cast<const int8_t*>(packed)[idx];
    }
}

// Minimal quantized attention kernel (Phase 2.2)
// Implements: output = softmax(Q @ K^T / sqrt(d)) @ V
// With on-the-fly dequantization of K and V

__global__ void quantized_attention_kernel_minimal(
    const float* __restrict__ query,
    const int8_t* __restrict__ key_int8,
    const float* __restrict__ key_scale,
    const int8_t* __restrict__ value_int8,
    const float* __restrict__ value_scale,
    const float* __restrict__ attention_mask,
    float* __restrict__ output,
    int B, int H, int q_len, int k_len, int d
) {
    // Grid: (B, H, q_len)
    int b = blockIdx.x;
    int h = blockIdx.y;
    int q_pos = blockIdx.z;

    if (b >= B || h >= H || q_pos >= q_len) return;

    // Shared memory layout:
    // [0..d-1]: query vector
    // [d..d+k_len-1]: attention scores
    // [d+k_len..d+k_len+31]: reduction scratch space
    extern __shared__ float shared_mem[];
    float* shared_query = shared_mem;
    float* shared_scores = shared_mem + d;
    float* shared_scratch = shared_mem + d + k_len;

    // Load query to shared memory
    for (int d_idx = threadIdx.x; d_idx < d; d_idx += blockDim.x) {
        int q_offset = ((b * H + h) * q_len + q_pos) * d + d_idx;
        shared_query[d_idx] = query[q_offset];
    }
    __syncthreads();

    float inv_sqrt_d = 1.0f / sqrtf((float)d);

    // Phase 1: Compute attention scores (Q @ K^T)
    for (int k_pos = threadIdx.x; k_pos < k_len; k_pos += blockDim.x) {
        float score = 0.0f;

        // Get scale for this key
        int k_scale_offset = (b * H + h) * k_len + k_pos;
        float k_scale_val = key_scale[k_scale_offset];

        // Dot product with on-the-fly dequantization
        for (int d_idx = 0; d_idx < d; d_idx++) {
            float q_val = shared_query[d_idx];

            int k_offset = ((b * H + h) * k_len + k_pos) * d + d_idx;
            int8_t k_q_val = key_int8[k_offset];
            float k_val = (float)k_q_val * k_scale_val;  // Fused dequantization

            score += q_val * k_val;
        }

        score *= inv_sqrt_d;

        // Apply attention mask if provided
        if (attention_mask != nullptr) {
            int mask_offset = ((b * 1 + 0) * q_len + q_pos) * k_len + k_pos;
            score += attention_mask[mask_offset];
        }

        shared_scores[k_pos] = score;
    }
    __syncthreads();

    // Phase 2: Numerically stable softmax
    // Find max score for numerical stability
    float max_score = -FLT_MAX;
    for (int k_pos = threadIdx.x; k_pos < k_len; k_pos += blockDim.x) {
        max_score = fmaxf(max_score, shared_scores[k_pos]);
    }
    max_score = block_reduce_max(max_score, shared_scratch);

    // Broadcast max to all threads
    if (threadIdx.x == 0) {
        shared_scratch[0] = max_score;
    }
    __syncthreads();
    max_score = shared_scratch[0];

    // Compute exp(score - max) and sum
    float exp_sum = 0.0f;
    for (int k_pos = threadIdx.x; k_pos < k_len; k_pos += blockDim.x) {
        float exp_score = expf(shared_scores[k_pos] - max_score);
        shared_scores[k_pos] = exp_score;
        exp_sum += exp_score;
    }
    exp_sum = block_reduce_sum(exp_sum, shared_scratch);

    // Broadcast sum to all threads
    if (threadIdx.x == 0) {
        shared_scratch[0] = exp_sum;
    }
    __syncthreads();
    exp_sum = shared_scratch[0];

    // Normalize to get probabilities
    for (int k_pos = threadIdx.x; k_pos < k_len; k_pos += blockDim.x) {
        shared_scores[k_pos] /= exp_sum;
    }
    __syncthreads();

    // Phase 3: Weighted sum of values (probs @ V)
    for (int d_idx = threadIdx.x; d_idx < d; d_idx += blockDim.x) {
        float output_val = 0.0f;

        for (int k_pos = 0; k_pos < k_len; k_pos++) {
            float prob = shared_scores[k_pos];

            // Get scale for this value
            int v_scale_offset = (b * H + h) * k_len + k_pos;
            float v_scale_val = value_scale[v_scale_offset];

            // Dequantize value and accumulate
            int v_offset = ((b * H + h) * k_len + k_pos) * d + d_idx;
            int8_t v_q_val = value_int8[v_offset];
            float v_val = (float)v_q_val * v_scale_val;

            output_val += prob * v_val;
        }

        // Write output
        int out_offset = ((b * H + h) * q_len + q_pos) * d + d_idx;
        output[out_offset] = output_val;
    }
}

} // namespace

// PyTorch wrapper
torch::Tensor quantized_attention_forward(
    torch::Tensor query,
    torch::Tensor key_int8,
    torch::Tensor key_scale,
    torch::Tensor value_int8,
    torch::Tensor value_scale,
    torch::optional<torch::Tensor> attention_mask
) {
    // Input validation
    TORCH_CHECK(query.is_cuda(), "query must be on CUDA");
    TORCH_CHECK(key_int8.is_cuda(), "key_int8 must be on CUDA");
    TORCH_CHECK(key_int8.dtype() == torch::kInt8, "key_int8 must be int8");
    TORCH_CHECK(value_int8.dtype() == torch::kInt8, "value_int8 must be int8");
    TORCH_CHECK(query.is_contiguous(), "query must be contiguous");

    const auto B = query.size(0);
    const auto H = query.size(1);
    const auto q_len = query.size(2);
    const auto d = query.size(3);
    const auto k_len = key_int8.size(2);

    // Allocate output
    auto output = torch::zeros_like(query);

    // Kernel launch configuration
    dim3 grid(B, H, q_len);
    int block_size = std::min(static_cast<int>(k_len), 256);
    dim3 block(block_size);

    // Shared memory: query + scores + scratch
    size_t shared_mem_size = (d + k_len + 32) * sizeof(float);

    // Get pointers
    const float* query_ptr = query.data_ptr<float>();
    const int8_t* key_int8_ptr = key_int8.data_ptr<int8_t>();
    const float* key_scale_ptr = key_scale.data_ptr<float>();
    const int8_t* value_int8_ptr = value_int8.data_ptr<int8_t>();
    const float* value_scale_ptr = value_scale.data_ptr<float>();
    const float* mask_ptr = attention_mask.has_value() ? attention_mask.value().data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();

    // Launch kernel
    quantized_attention_kernel_minimal<<<grid, block, shared_mem_size>>>(
        query_ptr,
        key_int8_ptr,
        key_scale_ptr,
        value_int8_ptr,
        value_scale_ptr,
        mask_ptr,
        output_ptr,
        B, H, q_len, k_len, d
    );

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        AT_ERROR("quantized_attention_kernel_minimal failed: ", hipGetErrorString(err));
    }

    return output;
}

__global__ void quantize_per_head_kernel(
    const float* __restrict__ input,
    int8_t* __restrict__ output,
    float* __restrict__ scales,
    int N,
    int H,
    int D,
    int bits,
    int max_q,
    int min_q
) {
    int n = blockIdx.x;
    int h = blockIdx.y;
    int idx = n * H + h;
    const float* row = input + idx * D;

    extern __shared__ float shared[];
    float* scratch = shared;  // size = blockDim.x / 32

    float local_max = 0.0f;
    for (int d = threadIdx.x; d < D; d += blockDim.x) {
        local_max = fmaxf(local_max, fabsf(row[d]));
    }
    local_max = block_reduce_max(local_max, scratch);

    __shared__ float scale;
    if (threadIdx.x == 0) {
        scale = local_max / static_cast<float>(max_q > 0 ? max_q : 1);
        if (scale == 0.0f) {
            scale = 1.0f;
        }
        scales[idx] = scale;
    }
    __syncthreads();

    float s = scale;
    int8_t* out_row = output + idx * D;
    for (int d = threadIdx.x; d < D; d += blockDim.x) {
        float q = roundf(row[d] / s);
        q = fmaxf(static_cast<float>(min_q), fminf(static_cast<float>(max_q), q));
        out_row[d] = static_cast<int8_t>(q);
    }
}

std::tuple<torch::Tensor, torch::Tensor> quantize_per_head_forward(
    torch::Tensor input,
    int bits
) {
    TORCH_CHECK(input.is_cuda(), "input must be on CUDA");
    TORCH_CHECK(input.dim() == 3, "input must have shape [N, H, D]");

    auto input_f32 = input.to(torch::kFloat32).contiguous();
    const int64_t N = input_f32.size(0);
    const int64_t H = input_f32.size(1);
    const int64_t D = input_f32.size(2);

    TORCH_CHECK(bits == 2 || bits == 3 || bits == 4 || bits == 8,
                "Unsupported bit-width for CUDA quantization");

    auto options_quant = torch::TensorOptions()
                             .dtype(torch::kInt8)
                             .device(input.device());
    auto options_scale = torch::TensorOptions()
                             .dtype(torch::kFloat32)
                             .device(input.device());

    auto quantized = torch::empty({N, H, D}, options_quant);
    auto scales = torch::empty({N, H}, options_scale);

    int max_q = (bits == 1) ? 0 : (1 << (bits - 1)) - 1;
    int min_q = (bits == 1) ? 0 : -(1 << (bits - 1));

    dim3 grid(N, H, 1);
    int threads = std::min<int64_t>(256, D);
    threads = std::max<int>(32, ((threads + 31) / 32) * 32);
    int num_warps = threads / 32;
    size_t shared_mem = static_cast<size_t>(num_warps) * sizeof(float);

    quantize_per_head_kernel<<<grid, threads, shared_mem>>>(
        input_f32.data_ptr<float>(),
        quantized.data_ptr<int8_t>(),
        scales.data_ptr<float>(),
        static_cast<int>(N),
        static_cast<int>(H),
        static_cast<int>(D),
        bits,
        max_q,
        min_q
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        AT_ERROR("quantize_per_head_kernel failed: ", hipGetErrorString(err));
    }

    return std::make_tuple(quantized, scales);
}

// ============================================================================
// Bucket-aware tiled attention kernel with streaming softmax
// ============================================================================

template<int BITS, int TILE_SIZE = 128>
__global__ void quantized_attention_bucket_tiled_kernel(
    const float* __restrict__ query,           // [B, H, q_len, d]
    const void* __restrict__ key_qx,           // [num_tokens, H, packed_dim] (uint8 if packed, int8 if not)
    const float* __restrict__ key_scale,       // [num_tokens, H]
    const void* __restrict__ value_qx,         // [num_tokens, H, packed_dim]
    const float* __restrict__ value_scale,     // [num_tokens, H]
    const float* __restrict__ attention_mask,  // [B, 1, q_len, full_k_len] or nullptr
    const int64_t* __restrict__ global_slots,  // [num_tokens] slot indices into full context
    float* __restrict__ output,                // [B, H, q_len, d] UNNORMALIZED
    float* __restrict__ m_out,                 // [B, H, q_len] max logit
    float* __restrict__ s_out,                 // [B, H, q_len] exp sum
    int B, int H, int q_len, int k_len, int d,
    int packed_dim,  // actual dimension of packed data (d for int8, packed size for 2/3/4-bit)
    bool is_packed,  // true if data is bit-packed
    int full_k_len,  // full context length for mask stride (may differ from k_len in bucket)
    int64_t key_stride_tokens,
    int64_t key_stride_heads,
    int64_t key_stride_dim,
    int64_t value_stride_tokens,
    int64_t value_stride_heads,
    int64_t value_stride_dim
) {
    // Grid: (B, H, q_len)
    int b = blockIdx.x;
    int h = blockIdx.y;
    int q_pos = blockIdx.z;

    if (b >= B || h >= H || q_pos >= q_len) return;

    // Shared memory layout:
    // [0..d-1]: query vector
    // [d..d+TILE_SIZE-1]: tile attention scores
    // [d+TILE_SIZE..d+TILE_SIZE+31]: reduction scratch
    // [d+TILE_SIZE+32..d+TILE_SIZE+32+d-1]: output accumulator (per-thread)
    extern __shared__ float shared_mem[];
    float* shared_query = shared_mem;
    float* shared_scores = shared_mem + d;
    float* shared_scratch = shared_mem + d + TILE_SIZE;
    float* output_acc = shared_mem + d + TILE_SIZE + 32;

    // Load query to shared memory
    for (int d_idx = threadIdx.x; d_idx < d; d_idx += blockDim.x) {
        int q_offset = ((b * H + h) * q_len + q_pos) * d + d_idx;
        shared_query[d_idx] = query[q_offset];
    }
    __syncthreads();

    float inv_sqrt_d = 1.0f / sqrtf((float)d);

    // Streaming softmax accumulators
    float m = -FLT_MAX;  // running max
    float s = 0.0f;      // running sum

    // Initialize output accumulator in shared memory
    for (int d_idx = threadIdx.x; d_idx < d; d_idx += blockDim.x) {
        output_acc[d_idx] = 0.0f;
    }

    // Number of tiles
    int num_tiles = (k_len + TILE_SIZE - 1) / TILE_SIZE;

    (void)packed_dim;  // kept for API compatibility

    const uint8_t* key_base_u8 = reinterpret_cast<const uint8_t*>(key_qx);
    const int8_t* key_base_i8 = reinterpret_cast<const int8_t*>(key_qx);
    const uint8_t* value_base_u8 = reinterpret_cast<const uint8_t*>(value_qx);
    const int8_t* value_base_i8 = reinterpret_cast<const int8_t*>(value_qx);

    // Process each tile
    for (int tile_idx = 0; tile_idx < num_tiles; tile_idx++) {
        int tile_start = tile_idx * TILE_SIZE;
        int tile_end = min(tile_start + TILE_SIZE, k_len);
        int tile_size = tile_end - tile_start;

        __syncthreads();

        // Phase 1: Compute attention scores for this tile (Q @ K^T)
        for (int local_k = threadIdx.x; local_k < tile_size; local_k += blockDim.x) {
            int k_pos = tile_start + local_k;
            float score = 0.0f;

            // Get scale for this key
            int k_scale_offset = k_pos * H + h;
            float k_scale_val = key_scale[k_scale_offset];

            // Compute dot product with on-the-fly unpacking
            int64_t key_row_offset = static_cast<int64_t>(k_pos) * key_stride_tokens +
                                     static_cast<int64_t>(h) * key_stride_heads;
            for (int d_idx = 0; d_idx < d; d_idx++) {
                float q_val = shared_query[d_idx];

                // Unpack key value
                int8_t k_q_val;
                if (is_packed && BITS < 8) {
                    // Packed: unpack from packed buffer
                    const uint8_t* k_packed_row = key_base_u8 + key_row_offset;
                    k_q_val = unpack_value<BITS>(k_packed_row, d_idx);
                } else {
                    // INT8 storage
                    int64_t k_offset = key_row_offset + static_cast<int64_t>(d_idx) * key_stride_dim;
                    k_q_val = key_base_i8[k_offset];
                }

                float k_val = (float)k_q_val * k_scale_val;
                score += q_val * k_val;
            }

            score *= inv_sqrt_d;

            // Apply attention mask if provided
            if (attention_mask != nullptr) {
                int slot = global_slots != nullptr ? static_cast<int>(global_slots[k_pos]) : k_pos;
                int mask_offset = ((b * 1 + 0) * q_len + q_pos) * full_k_len + slot;
                score += attention_mask[mask_offset];
            }

            shared_scores[local_k] = score;
        }
        __syncthreads();

        // Phase 2: Streaming softmax update
        // Find max score in this tile
        float tile_max = -FLT_MAX;
        for (int local_k = threadIdx.x; local_k < tile_size; local_k += blockDim.x) {
            tile_max = fmaxf(tile_max, shared_scores[local_k]);
        }
        tile_max = block_reduce_max(tile_max, shared_scratch);
        __syncthreads();

        // Broadcast tile_max
        if (threadIdx.x == 0) {
            shared_scratch[0] = tile_max;
        }
        __syncthreads();
        tile_max = shared_scratch[0];

        // Update global max and rescale previous accumulator
        float m_prev = m;
        m = fmaxf(m, tile_max);
        float rescale_factor = expf(m_prev - m);

        // Rescale output accumulator from previous tiles
        for (int d_idx = threadIdx.x; d_idx < d; d_idx += blockDim.x) {
            output_acc[d_idx] *= rescale_factor;
        }

        // Compute exp(score - m) and sum for this tile
        float tile_sum = 0.0f;
        for (int local_k = threadIdx.x; local_k < tile_size; local_k += blockDim.x) {
            float exp_score = expf(shared_scores[local_k] - m);
            shared_scores[local_k] = exp_score;
            tile_sum += exp_score;
        }
        tile_sum = block_reduce_sum(tile_sum, shared_scratch);
        __syncthreads();

        // Broadcast tile_sum
        if (threadIdx.x == 0) {
            shared_scratch[0] = tile_sum;
        }
        __syncthreads();
        tile_sum = shared_scratch[0];

        // Update global sum
        s = s * rescale_factor + tile_sum;

        // Phase 3: Accumulate attention @ V for this tile
        __syncthreads();
        for (int d_idx = threadIdx.x; d_idx < d; d_idx += blockDim.x) {
            float v_acc = 0.0f;
            for (int local_k = 0; local_k < tile_size; local_k++) {
                int k_pos = tile_start + local_k;
                float prob = shared_scores[local_k];

                // Get scale for this value
                int v_scale_offset = k_pos * H + h;
                float v_scale_val = value_scale[v_scale_offset];

                // Unpack value
                int8_t v_q_val;
                if (is_packed && BITS < 8) {
                    // Packed: unpack from packed buffer
                    int64_t value_row_offset = static_cast<int64_t>(k_pos) * value_stride_tokens +
                                              static_cast<int64_t>(h) * value_stride_heads;
                    const uint8_t* v_packed_row = value_base_u8 + value_row_offset;
                    v_q_val = unpack_value<BITS>(v_packed_row, d_idx);
                } else {
                    // INT8 storage
                    int64_t value_row_offset = static_cast<int64_t>(k_pos) * value_stride_tokens +
                                              static_cast<int64_t>(h) * value_stride_heads +
                                              static_cast<int64_t>(d_idx) * value_stride_dim;
                    v_q_val = value_base_i8[value_row_offset];
                }

                float v_val = (float)v_q_val * v_scale_val;
                v_acc += prob * v_val;
            }

            output_acc[d_idx] += v_acc;
        }
    }

    // Write unnormalized output and softmax statistics
    __syncthreads();
    for (int d_idx = threadIdx.x; d_idx < d; d_idx += blockDim.x) {
        int out_offset = ((b * H + h) * q_len + q_pos) * d + d_idx;
        output[out_offset] = output_acc[d_idx];  // UNNORMALIZED for cross-bucket accumulation
    }

    // Thread 0 writes softmax statistics
    if (threadIdx.x == 0) {
        int stats_offset = (b * H + h) * q_len + q_pos;
        m_out[stats_offset] = m;
        s_out[stats_offset] = s;
    }
}

// Helper to launch bucket kernel with appropriate template instantiation
template<int TILE_SIZE = 128>
void launch_bucket_kernel(
    int bits,
    torch::Tensor query,
    torch::Tensor key_qx,
    torch::Tensor key_scale,
    torch::Tensor value_qx,
    torch::Tensor value_scale,
    torch::optional<torch::Tensor> attention_mask,
    torch::Tensor global_slots,
    torch::Tensor output,
    torch::Tensor m_out,
    torch::Tensor s_out,
    int packed_dim,
    bool is_packed,
    int full_k_len
) {
    const auto B = query.size(0);
    const auto H = query.size(1);
    const auto q_len = query.size(2);
    const auto d = query.size(3);
    const auto k_len = key_qx.size(0);

    // Kernel launch configuration
    dim3 grid(B, H, q_len);
    int block_size = std::min(256, std::max(32, (int)d));
    dim3 block(block_size);

    // Shared memory: query + tile scores + scratch + output accumulator
    size_t shared_mem_size = (d + TILE_SIZE + 32 + d) * sizeof(float);

    const float* query_ptr = query.data_ptr<float>();
    const void* key_qx_ptr = key_qx.data_ptr();
    const float* key_scale_ptr = key_scale.data_ptr<float>();
    const void* value_qx_ptr = value_qx.data_ptr();
    const float* value_scale_ptr = value_scale.data_ptr<float>();
    const float* mask_ptr = attention_mask.has_value() ? attention_mask.value().data_ptr<float>() : nullptr;
    float* output_ptr = output.data_ptr<float>();
    const int64_t* slots_ptr = global_slots.defined() ? global_slots.data_ptr<int64_t>() : nullptr;
    float* m_ptr = m_out.data_ptr<float>();
    float* s_ptr = s_out.data_ptr<float>();

    // Extract strides (in elements) for packed/int8 layouts
    auto key_strides = key_qx.strides();
    auto value_strides = value_qx.strides();

    int64_t key_stride_tokens = key_strides.size() > 0 ? key_strides[0] : 0;
    int64_t key_stride_heads = key_strides.size() > 1 ? key_strides[1] : 0;
    int64_t key_stride_dim = key_strides.size() > 2 ? key_strides[2] : 1;

    int64_t value_stride_tokens = value_strides.size() > 0 ? value_strides[0] : 0;
    int64_t value_stride_heads = value_strides.size() > 1 ? value_strides[1] : 0;
    int64_t value_stride_dim = value_strides.size() > 2 ? value_strides[2] : 1;

    // Dispatch to appropriate template based on bits
    if (bits == 2) {
        quantized_attention_bucket_tiled_kernel<2, TILE_SIZE><<<grid, block, shared_mem_size>>>(
            query_ptr, key_qx_ptr, key_scale_ptr, value_qx_ptr, value_scale_ptr,
            mask_ptr, slots_ptr, output_ptr, m_ptr, s_ptr, B, H, q_len, k_len, d,
            packed_dim, is_packed, full_k_len,
            key_stride_tokens, key_stride_heads, key_stride_dim,
            value_stride_tokens, value_stride_heads, value_stride_dim
        );
    } else if (bits == 3) {
        quantized_attention_bucket_tiled_kernel<3, TILE_SIZE><<<grid, block, shared_mem_size>>>(
            query_ptr, key_qx_ptr, key_scale_ptr, value_qx_ptr, value_scale_ptr,
            mask_ptr, slots_ptr, output_ptr, m_ptr, s_ptr, B, H, q_len, k_len, d,
            packed_dim, is_packed, full_k_len,
            key_stride_tokens, key_stride_heads, key_stride_dim,
            value_stride_tokens, value_stride_heads, value_stride_dim
        );
    } else if (bits == 4) {
        quantized_attention_bucket_tiled_kernel<4, TILE_SIZE><<<grid, block, shared_mem_size>>>(
            query_ptr, key_qx_ptr, key_scale_ptr, value_qx_ptr, value_scale_ptr,
            mask_ptr, slots_ptr, output_ptr, m_ptr, s_ptr, B, H, q_len, k_len, d,
            packed_dim, is_packed, full_k_len,
            key_stride_tokens, key_stride_heads, key_stride_dim,
            value_stride_tokens, value_stride_heads, value_stride_dim
        );
    } else if (bits == 8) {
        quantized_attention_bucket_tiled_kernel<8, TILE_SIZE><<<grid, block, shared_mem_size>>>(
            query_ptr, key_qx_ptr, key_scale_ptr, value_qx_ptr, value_scale_ptr,
            mask_ptr, slots_ptr, output_ptr, m_ptr, s_ptr, B, H, q_len, k_len, d,
            packed_dim, is_packed, full_k_len,
            key_stride_tokens, key_stride_heads, key_stride_dim,
            value_stride_tokens, value_stride_heads, value_stride_dim
        );
    } else {
        AT_ERROR("Unsupported bit-width: ", bits);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        AT_ERROR("quantized_attention_bucket_tiled_kernel failed: ", hipGetErrorString(err));
    }
}

// PyTorch wrapper for bucketed attention (returns unnormalized output + statistics)
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> quantized_attention_bucket_forward(
    torch::Tensor query,
    torch::Tensor key_qx,
    torch::Tensor key_scale,
    torch::Tensor value_qx,
    torch::Tensor value_scale,
    torch::Tensor global_slots,
    int bits,
    int packed_dim,
    bool is_packed,
    torch::optional<torch::Tensor> attention_mask,
    int full_k_len
) {
    // Input validation
    TORCH_CHECK(query.is_cuda(), "query must be on CUDA");
    TORCH_CHECK(key_qx.is_cuda(), "key_qx must be on CUDA");
    TORCH_CHECK(query.is_contiguous(), "query must be contiguous");
    TORCH_CHECK(key_qx.is_contiguous(), "key_qx must be contiguous");
    TORCH_CHECK(key_scale.is_contiguous(), "key_scale must be contiguous");
    TORCH_CHECK(value_qx.is_contiguous(), "value_qx must be contiguous");
    TORCH_CHECK(value_scale.is_contiguous(), "value_scale must be contiguous");
    TORCH_CHECK(global_slots.dtype() == torch::kLong, "global_slots must be int64 tensor");
    TORCH_CHECK(global_slots.is_contiguous(), "global_slots must be contiguous");

    const auto B = query.size(0);
    const auto H = query.size(1);
    const auto q_len = query.size(2);
    const auto d = query.size(3);
    const auto k_len = key_qx.size(0);
    TORCH_CHECK(global_slots.numel() == k_len,
                "global_slots must have same length as bucket tokens");

    // If full_k_len not provided, default to bucket size
    if (full_k_len <= 0) {
        full_k_len = k_len;
    }

    // Allocate output (unnormalized) and softmax statistics
    auto output = torch::zeros_like(query);
    auto m_out = torch::zeros({B, H, q_len}, query.options());
    auto s_out = torch::zeros({B, H, q_len}, query.options());

    // Launch kernel for this bucket
    launch_bucket_kernel(bits, query, key_qx, key_scale, value_qx, value_scale,
                         attention_mask, global_slots, output, m_out, s_out,
                         packed_dim, is_packed, full_k_len);

    return std::make_tuple(output, m_out, s_out);
}
